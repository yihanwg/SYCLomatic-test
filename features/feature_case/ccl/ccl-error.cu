// ====------ ccl-error.cu---------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include "nccl.h"

int main(){
  int version;
  ncclResult_t res;
  res = ncclGetVersion(&version);
  ncclGetErrorString(res);
  ncclGetLastError(NULL);
  if (res == ncclSuccess) {
    return 0;
  }
}